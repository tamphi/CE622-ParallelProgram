#include "hip/hip_runtime.h"
/*
 *  Host-side code for Gaussian elimination. 
 * 
 * Author: Naga Kandasamy
 * Date modified: March 2, 2021
 * 
 * Student name(s): FIXME
 * Date modified: FIXME
*/

#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>

#include "gauss_eliminate_kernel.cu"

#define MIN_NUMBER 2
#define MAX_NUMBER 50

extern "C" int compute_gold(float*, const float*, unsigned int);
Matrix allocate_matrix_on_gpu(const Matrix M);
Matrix allocate_matrix(int num_rows, int num_columns, int init);
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost);
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice);
void gauss_eliminate_on_device(const Matrix M, Matrix P);
int perform_simple_check(const Matrix M);
void print_matrix(const Matrix M);
void write_matrix_to_file(const Matrix M);
float get_random_number(int, int);
void check_CUDA_error(const char *msg);
int check_results(float *reference, float *gpu_result, int num_elements, float threshold);


int main(int argc, char** argv) 
{
    if (argc > 1) {
        printf("Error. This program accepts no arguments.\n");
        exit(EXIT_SUCCESS);
    }
	
    Matrix  A; /* The N x N input matrix */
	Matrix  U; /* The upper triangular matrix returned by device */ 
	
	/* Allocate and initialize the matrices */
    srand(time(NULL));
	A  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 1);
	U  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0); 

	/* Perform Gaussian elimination on the CPU */
	Matrix reference = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0);

    //printf("\nMatrix ORIGINAL:\n");
    //print_matrix(reference);
    struct timeval start, stop;	
	gettimeofday(&start, NULL);

	    int status = compute_gold(reference.elements, A.elements, A.num_rows);

    gettimeofday(&stop, NULL);
	printf("CPU time: %f\n",stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000);

	if (status == 0) { 
		printf("Failed to convert given matrix to upper triangular. Try again. Exiting. \n");
		exit(EXIT_FAILURE);
	}
	
    status = perform_simple_check(reference); // Check that the principal diagonal elements are 1 
	if (status == 0) {
		printf("The upper triangular matrix is incorrect. Exiting. \n");
		exit(EXIT_FAILURE); 
	}
	printf("MATRIX SIZE: %d\n", MATRIX_SIZE);
    printf("Gaussian elimination on the CPU was successful. \n");
#ifdef DEBUG
    printf("\nMatrix GOLD:\n");
    print_matrix(reference);
#endif
	/* Perform Gaussin elimination on device. Return the result in U. */
	gauss_eliminate_on_device(A, U);
    
	/* Check if device result matches reference. */
	int num_elements = MATRIX_SIZE*MATRIX_SIZE;
    int res = check_results(reference.elements, U.elements, num_elements, 0.001f);
    printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

	/* Free host matrices. */
	free(A.elements); 
	free(U.elements); 
	free(reference.elements);

    exit(EXIT_SUCCESS);
}

/* FIXME: complete this function. */
void gauss_eliminate_on_device(const Matrix A, Matrix U)
{
    
    struct timeval start, stop;	

    Matrix device_U = allocate_matrix_on_gpu(A);
    copy_matrix_to_device(device_U,A);
    //dim3 block(256,1,1);
    dim3 block(THREAD_SIZE,1,1);
    dim3 grid(THREAD_SIZE,1);

    gettimeofday(&start, NULL);

    int size = MATRIX_SIZE;
    for(int k = 0; k < size; k++){
       division__kernel<<<grid, block>>>(device_U.elements, k, size);
       elimination_kernel<<<grid, block>>>(device_U.elements,k,size);
    }
    hipDeviceSynchronize();
    check_CUDA_error("KERNEL FAILURE");

    gettimeofday(&stop, NULL);
    printf("GPU time: %f\n",stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000);

    copy_matrix_from_device(U, device_U);
#ifdef DEBUG
    printf("\nMatrix U:\n");
    print_matrix(U);
#endif
    hipFree(&device_U);
}

/* Allocate device matrix of same size as M. */
Matrix allocate_matrix_on_gpu(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

/* Allocate matrix of dimensions height * width
   If init == 0, initialize to all zeroes.  
   If init == 1, perform random initialization.
*/
Matrix allocate_matrix(int num_rows, int num_columns, int init)
{
    Matrix M;
    M.num_columns = M.pitch = num_columns;
    M.num_rows = num_rows;
    int size = M.num_rows * M.num_columns;
		
	M.elements = (float*)malloc(size*sizeof(float));
	for (unsigned int i = 0; i < size; i++) {
		if (init == 0) 
            M.elements[i] = 0; 
		else
            M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}
    
    return M;
}	

/* Copy matrix to from host to device */
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
}

/* Copy matrix from device to host */
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

/* Print matrix to screen */
void print_matrix(const Matrix M)
{
	for (unsigned int i = 0; i < M.num_rows; i++){
		for (unsigned int j = 0; j < M.num_columns; j++)
			printf("%f ", M.elements[i*M.num_rows + j]);
		printf("\n");
	} 
	printf("\n");
}

/* Return a random number between [min, max] */ 
float get_random_number(int min, int max)
{
	return (float)floor((double)(min + (max - min + 1)*((float)rand()/(float)RAND_MAX)));
}

/* Check to see if the principal diagonal elements are 1 */
int perform_simple_check(const Matrix M)
{
	for (unsigned int i = 0; i < M.num_rows; i++)
        if ((fabs(M.elements[M.num_rows*i + i] - 1.0)) > 0.001) return 0;
	
    return 1;
} 

void check_CUDA_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) 
	{
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}						 
}

int check_results(float *reference, float *gpu_result, int num_elements, float threshold)
{

    int i;
    int check = 1;
    float epsilon = 0.0;

    for (i = 0; i < num_elements; i++){
        if (fabsf(reference[i] - gpu_result[i]) > threshold) {
           check = 0;
            break;
        }
    }

    for (i = 0; i < num_elements; i++){
       if (fabsf(reference[i] - gpu_result[i]) > epsilon) {
           epsilon = fabsf(reference[i] - gpu_result[i]);
       }
    }

    printf("Max epsilon = %f. \n", epsilon);
    return check;
}
